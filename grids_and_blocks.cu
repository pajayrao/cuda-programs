/*
 * Check grid and block dimensions
 */


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void checkIndex(void)
{
	printf("threadIdx : (%d,%d,%d) blockIdx : (%d,%d,%d)) blockDim : (%d,%d,%d) gridDim : (%d,%d,%d)\n ",threadIdx.x,threadIdx.y,threadIdx.z,blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,blockDim.z,gridDim.x,gridDim.y,gridDim.z);
}

int main(void)
{	int n=6;
	dim3 block(3);
	dim3 grid((n+block.x-1)/block.x);

	checkIndex<<<grid,block>>>();
	hipDeviceReset();

}
