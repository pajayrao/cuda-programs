/*
 * Concurrent Kernels Execution
 */


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernel_1()
{
	double sum = 0.0;
	for(int i=0;i<10;i++){
		sum=sum+tan(0.1)*tan(0.1);

	}
}

__global__ void kernel_2()
{
	double sum = 0.0;
	for(int i=0;i<20;i++){
		sum=sum+tan(0.2)*tan(0.2);

	}
}

__global__ void kernel_3()
{
	double sum = 0.0;
	for(int i=0;i<30;i++){
		sum=sum+tan(0.3)*tan(0.3);

	}
}

__global__ void kernel_4()
{
	double sum = 0.0;
	for(int i=0;i<40;i++){
		sum=sum+tan(0.4)*tan(0.4);

	}
}

int main()
{
	int n_streams=5,i,j;
	hipStream_t *streams = (hipStream_t*)malloc(n_streams*sizeof(hipStream_t));

	for(i=0;i<n_streams;i++)
	{
		hipStreamCreate(&streams[i]);

	}
  
  dim3 block(1);
	dim3 grid(1);

	for(i=0;i<n_streams;i++)
	{
		kernel_1<<<grid,block,0,streams[i]>>>();
		kernel_2<<<grid,block,0,streams[i]>>>();
		kernel_3<<<grid,block,0,streams[i]>>>();
		kernel_4<<<grid,block,0,streams[i]>>>();

	}


	return 0;
}
