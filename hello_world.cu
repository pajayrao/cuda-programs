/*
 * Hello World Program from GPU
 */



#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void helloWorldFromGPU(void)
{	int x= threadIdx.x;
	printf("Hello World from GPU! thread id  %d\n",x);

}

int main(void)
{
	printf("Hello World from CPU!");
	helloWorldFromGPU<<<1,10>>>();
	hipDeviceSynchronize();
	return 0;

}
